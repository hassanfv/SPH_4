#include "hip/hip_runtime.h"
%%writefile stiff_test.cu
#include <iostream>
#include <cstdio>
#include "diff_jacob_batch_01.h"
#include "stiff_libs_hfv_batch_01.h"
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

int main()
{
  const int N_part = 100000; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  const int N_con = 5000; //!!!!!!!!!!! number of concurrent access of the matrices a and dfdy !// Select efficiently. The higher the better !
  int *h_slot_status = new int[N_con];
  for (int i = 0; i < N_con; i++)
    h_slot_status[i] = 0;
  
  const int n = 3; // !!!!!!!!!!!!!!!!!! This is nvar !!!!!!!!!!!!!!!!! Also modify the value for vv in ludcmp function !!!!!!

  const float x1 = 0.0f; // Initial time !!!!!!!!!!!!!!!!!!!!!!!
	const float x2 = 50.0f; // Final time !!!!!!!!!!!!!!!!!!!!!!!!!
	float eps = 1e-4f;   // THIS affects the execution time !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	float htry = 2.9e-4f; // Step size !!!!!!!!!!!!!!!!!!!!!!!!
	float hmin = 1e-6f;  //!!!!!!!!!!!!!!!!!!!!!!!!!! To be adjusted for each problem !!!!!!!!!!!!!!!!!!!!!!!!!!

  float *h_y0 = new float[N_part * n];
  //!!!!!!!!!!!!! initial values !!!!!!!!!!!!!!!!!
  // These will be the initial ionization fractions of each SPH particle!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < N_part; i++)
  {
    h_y0[i * n + 0] = 1.0f; // e.g. 1.0f will be replaced by nHII or nHeI of particle i !
    h_y0[i * n + 1] = 1.0f;
    h_y0[i * n + 2] = 0.0f;
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!! 

  // Allocate device memory-- except "d_y0", all other device variables are only needed in the device and they do not need host counterparts !
  float *d_y0, *d_dydx, *d_yscal; // Note that the final result will also be copied back to d_y0 !
  float *d_a, *d_dfdy, *d_dfdx, *d_dysav, *d_err, *d_ysav; // a and dfdy are each a matrix !
  float *d_g1, *d_g2, *d_g3, *d_g4;
  int *d_indx, *d_slot_status;

  CHECK_CUDA(hipMalloc(&d_slot_status, N_con * sizeof(int)));

  CHECK_CUDA(hipMalloc(&d_y0, N_part * n * sizeof(float))); // Only y0 has N_part*n others have N_con*n.
  CHECK_CUDA(hipMalloc(&d_dydx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_yscal, N_con * n * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_a, N_con * n * n * sizeof(float)));  // 3D !
  CHECK_CUDA(hipMalloc(&d_dfdy, N_con * n * n * sizeof(float))); // 3D !
  CHECK_CUDA(hipMalloc(&d_indx, N_con * n * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_dfdx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_dysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_err, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_ysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g1, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g2, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g3, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g4, N_con * n * sizeof(float)));

  // Copy to device
  CHECK_CUDA(hipMemcpy(d_slot_status, h_slot_status, N_con * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_y0, h_y0, N_part * n * sizeof(float), hipMemcpyHostToDevice));



  int nThreadsPerBlock = 256;
  //int nBlocksPerGrid = (N_part + nThreadsPerBlock - 1) / nThreadsPerBlock; // automate it !!
  
  auto start = chrono::high_resolution_clock::now();
  
  //************** Batch running of the Kernel ****************
  for (int i = 0; i < N_part; i += N_con)
  {
    int N_batch = min(N_con, N_part - i); // last batch might be smaller.
    
    odeint_d<<<(N_batch + nThreadsPerBlock - 1)/nThreadsPerBlock, nThreadsPerBlock>>>(d_y0 + i * n, x1, x2, eps, htry, hmin, n, d_dydx, d_yscal, d_a,
                                                                                      d_dfdy, d_indx, d_dfdx, d_dysav, d_err, d_ysav, d_g1, d_g2, d_g3,
                                                                                      d_g4, d_slot_status, N_batch); // n is nvar !
  }
  //***********************************************************
  hipDeviceSynchronize();
  
  auto end = chrono::high_resolution_clock::now();
  chrono::duration<double> elapsed = end - start;
  cout << "Elapsed time: " << elapsed.count() << " seconds" << endl;
  
  hipMemcpy(h_y0, d_y0, N_part * n * sizeof(float), hipMemcpyDeviceToHost);
  
  int i = 0; // i is an SPH particle index.
  
  float y0 = h_y0[i * n + 0];
  float y1 = h_y0[i * n + 1];
  float y2 = h_y0[i * n + 2];
  
  cout << y0 << ", " << y1 << ", " << y2 << endl;
  
}


