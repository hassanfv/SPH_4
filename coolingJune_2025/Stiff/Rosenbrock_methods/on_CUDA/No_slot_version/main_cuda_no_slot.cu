#include "hip/hip_runtime.h"
%%writefile stiff_test.cu
#include <iostream>
#include <cstdio>
#include "stiff_libs_hfv_no_slot_01.h"
#include <hip/hip_runtime.h>
#include <chrono>

using namespace std;

int main()
{

  const int N_part = 1000000; //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  unsigned long long* d_timings;
  hipMalloc(&d_timings, N_part * sizeof(unsigned long long));

  const int N_con = N_part; //!!!!!!!!!!! number of concurrent access of the matrices a and dfdy !// Select efficiently. The higher the better !
  int *h_slot_status = new int[N_con];
  for (int i = 0; i < N_con; i++)
    h_slot_status[i] = 0;

  const int n = 3; // !!!!!!!!!!!!!!!!!! This is nvar !!!!!!!!!!!!!!!!! Also modify the value for vv in ludcmp function !!!!!!

  const float x1 = 0.0f; // Initial time !!!!!!!!!!!!!!!!!!!!!!!
	const float x2 = 50.0f; // Final time !!!!!!!!!!!!!!!!!!!!!!!!!
	float eps = 1e-4f;   // THIS affects the execution time !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	float htry = 2.9e-4f; // Step size !!!!!!!!!!!!!!!!!!!!!!!!
	float hmin = 1e-6f;  //!!!!!!!!!!!!!!!!!!!!!!!!!! To be adjusted for each problem !!!!!!!!!!!!!!!!!!!!!!!!!!

  float *h_y0 = new float[N_part * n];
  //!!!!!!!!!!!!! initial values !!!!!!!!!!!!!!!!!
  // These will be the initial ionization fractions of each SPH particle!
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
  for (int i = 0; i < N_part; i++)
  {
    h_y0[i * n + 0] = 1.0f; // e.g. 1.0f will be replaced by nHII or nHeI of particle i !
    h_y0[i * n + 1] = 1.0f;
    h_y0[i * n + 2] = 0.0f;
  }
  //!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

  // Allocate device memory-- except "d_y0", all other device variables are only needed in the device and they do not need host counterparts !
  float *d_y0, *d_dydx, *d_yscal; // Note that the final result will also be copied back to d_y0 !
  float *d_a, *d_dfdy, *d_dfdx, *d_dysav, *d_err, *d_ysav; // a and dfdy are each a matrix !
  float *d_g1, *d_g2, *d_g3, *d_g4;
  int *d_indx, *d_slot_status;

  CHECK_CUDA(hipMalloc(&d_slot_status, N_con * sizeof(int)));

  CHECK_CUDA(hipMalloc(&d_y0, N_part * n * sizeof(float))); // Only y0 has N_part*n others have N_con*n.
  CHECK_CUDA(hipMalloc(&d_dydx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_yscal, N_con * n * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_a, N_con * n * n * sizeof(float)));  // 3D !
  CHECK_CUDA(hipMalloc(&d_dfdy, N_con * n * n * sizeof(float))); // 3D !
  CHECK_CUDA(hipMalloc(&d_indx, N_con * n * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_dfdx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_dysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_err, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_ysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g1, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g2, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g3, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g4, N_con * n * sizeof(float)));

  // Copy to device
  CHECK_CUDA(hipMemcpy(d_slot_status, h_slot_status, N_con * sizeof(int), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_y0, h_y0, N_part * n * sizeof(float), hipMemcpyHostToDevice));

  int nThreadsPerBlock = 512;
  int nBlocksPerGrid = (N_part + nThreadsPerBlock - 1) / nThreadsPerBlock; // automate it !!


  size_t free_mem_before, total_mem;
  hipMemGetInfo(&free_mem_before, &total_mem);
  std::cout << "[Before kernel] Free: " << free_mem_before / (1024.0 * 1024.0) << " MB, "
            << "Used: " << (total_mem - free_mem_before) / (1024.0 * 1024.0) << " MB"
            << " / " << total_mem / (1024.0 * 1024.0) << " MB total\n";


  auto start = chrono::high_resolution_clock::now();
  // the final evolved y will be overwritten in d_y0 !
  odeint_d<<<nBlocksPerGrid, nThreadsPerBlock>>>(d_y0, x1, x2, eps, htry, hmin, n, d_dydx, d_yscal, d_a, d_dfdy, d_indx, d_dfdx, d_dysav,
                                                 d_err, d_ysav, d_g1, d_g2, d_g3, d_g4, d_slot_status, N_con, N_part, d_timings); // n is nvar !
  hipDeviceSynchronize();
  auto end = chrono::high_resolution_clock::now();
  chrono::duration<double> elapsed = end - start;
  cout << "Elapsed time: " << elapsed.count() << " seconds" << endl;
  
  
  size_t free_mem_after;
  hipMemGetInfo(&free_mem_after, &total_mem);
  std::cout << "[After kernel] Free: " << free_mem_after / (1024.0 * 1024.0) << " MB, "
            << "Used: " << (total_mem - free_mem_after) / (1024.0 * 1024.0) << " MB"
            << " / " << total_mem / (1024.0 * 1024.0) << " MB total\n";


  hipMemcpy(h_y0, d_y0, N_part * n * sizeof(float), hipMemcpyDeviceToHost);

  int i = 1000; // i is an SPH particle index.

  float y0 = h_y0[i * n + 0];
  float y1 = h_y0[i * n + 1];
  float y2 = h_y0[i * n + 2];

  cout << y0 << ", " << y1 << ", " << y2 << endl;



  unsigned long long* h_timings = new unsigned long long[N_part];
  hipMemcpy(h_timings, d_timings, N_part * sizeof(unsigned long long), hipMemcpyDeviceToHost);

  // Example: Print timing of first 10 threads
  for (int i = 0; i < 10; ++i)
    std::cout << "Thread " << i << " took " << h_timings[i] << " cycles" << std::endl;


}


