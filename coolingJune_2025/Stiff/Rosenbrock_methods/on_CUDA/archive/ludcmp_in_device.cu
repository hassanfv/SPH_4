
#include <hip/hip_runtime.h>



__device__ void ludcmp(float *a, int *indx, int n)
{
    const float TINY = 1.0e-20f;
    int i, imax = 0, j, k;
    float big, dum, sum, temp;

    // Allocate vv on stack (private to thread)
    float vv[3];  // change 3 to appropriate value !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!

    float d = 1.0f;

    for (i = 0; i < n; i++) 
    {
        big = 0.0f;
        for (j = 0; j < n; j++) 
        {
            temp = fabsf(a[i * n + j]);
            if (temp > big) big = temp;
        }
        if (big == 0.0f) 
        {
            // No printf in production device code, consider setting a flag or return NaN later!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
            return;
        }
        vv[i] = 1.0f / big;
    }

    for (j = 0; j < n; j++) 
    {
        for (i = 0; i < j; i++) 
        {
            sum = a[i * n + j];
            for (k = 0; k < i; k++)
                sum -= a[i * n + k] * a[k * n + j];
            a[i * n + j] = sum;
        }

        big = 0.0f;
        for (i = j; i < n; i++) 
        {
            sum = a[i * n + j];
            for (k = 0; k < j; k++)
                sum -= a[i * n + k] * a[k * n + j];
            a[i * n + j] = sum;
            dum = vv[i] * fabsf(sum);
            if (dum >= big) 
            {
                big = dum;
                imax = i;
            }
        }

        if (j != imax) 
        {
            for (k = 0; k < n; k++) 
            {
                dum = a[imax * n + k];
                a[imax * n + k] = a[j * n + k];
                a[j * n + k] = dum;
            }
            d = -d;
            vv[imax] = vv[j];
        }

        indx[j] = imax;
        if (a[j * n + j] == 0.0f) a[j * n + j] = TINY;

        if (j != n - 1) 
        {
            dum = 1.0f / a[j * n + j];
            for (i = j + 1; i < n; i++)
                a[i * n + j] *= dum;
        }
    }
}





