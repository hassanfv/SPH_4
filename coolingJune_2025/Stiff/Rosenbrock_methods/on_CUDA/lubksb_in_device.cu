
#include <hip/hip_runtime.h>



__device__ void lubksb(float *a, int *indx, float *b, int n)
{
    int i, ii = 0, ip, j;
    float sum;

    for (i = 0; i < n; i++) 
    {
        ip = indx[i];
        sum = b[ip];
        b[ip] = b[i];

        if (ii != 0) {
            for (j = ii - 1; j < i; j++) 
            {
                sum -= a[i * n + j] * b[j];
            }
        } 
        else if (sum != 0.0f) 
        {
            ii = i + 1;
        }

        b[i] = sum;
    }

    for (i = n - 1; i >= 0; i--) 
    {
        sum = b[i];
        for (j = i + 1; j < n; j++) 
        {
            sum -= a[i * n + j] * b[j];
        }
        b[i] = sum / a[i * n + i];
    }
}

