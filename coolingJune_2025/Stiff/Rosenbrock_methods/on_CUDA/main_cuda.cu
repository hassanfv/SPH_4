



N_con *


int main()
{

  const float x1 = 0.0f; // Initial time !!!!!!!!!!!!!!!!!!!!!!!
	const float x2 = 50.0f; // Final time !!!!!!!!!!!!!!!!!!!!!!!!!
	float eps = 1e-4f;   // THIS affects the execution time !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	float htry = 2.9e-4f; // Step size !!!!!!!!!!!!!!!!!!!!!!!!
	float hmin = 1e-6f;  //!!!!!!!!!!!!!!!!!!!!!!!!!! To be adjusted for each problem !!!!!!!!!!!!!!!!!!!!!!!!!!

  float y_host[N] = {1.0f, 0.0f, 0.0f, 0.0f};  // initial values
  float dydx_host[N], yscal_host[N], hdid_host, hnext_host;


  // Allocate device memory
  float *d_y, *d_dydx, *d_x, *d_yscal, *d_hdid, *d_hnext;
  float *d_a, *d_dfdy, *d_dfdx, *d_dysav, *d_err, *d_ysav;
  float *d_g1, *d_g2, *d_g3, *d_g4;
  int *d_indx;

  CHECK_CUDA(hipMalloc(&d_x, N_con * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_hdid, N_con * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_hnext, N_con * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_y, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_dydx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_yscal, N_con * n * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_a, N_con * n * n * sizeof(float)));  // 3D !
  CHECK_CUDA(hipMalloc(&d_dfdy, N_con * n * n * sizeof(float))); // 3D !
  CHECK_CUDA(hipMalloc(&d_indx, N_con * n * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_dfdx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_dysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_err, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_ysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g1, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g2, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g3, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g4, N_con * n * sizeof(float)));

  // Copy inputs to device
  CHECK_CUDA(hipMemcpy(d_x, &h_x, N_con * sizeof(float), hipMemcpyHostToDevice));
  
  CHECK_CUDA(hipMemcpy(d_y, h_y, N_con * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_dydx, h_dydx, N_con * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_yscal, h_yscal, N_con * n * sizeof(float), hipMemcpyHostToDevice));
  
//!!!!!!!!!!!!!!! TRY TO GET RID of h_x and d_x

}
