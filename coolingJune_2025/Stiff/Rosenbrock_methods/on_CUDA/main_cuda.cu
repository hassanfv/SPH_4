



N_con *


int main()
{

  // Allocate device memory
  float *d_y, *d_dydx, *d_x, *d_yscal, *d_hdid, *d_hnext;
  float *d_a, *d_dfdy, *d_dfdx, *d_dysav, *d_err, *d_ysav;
  float *d_g1, *d_g2, *d_g3, *d_g4;
  int *d_indx;

  CHECK_CUDA(hipMalloc(&d_x, N_con * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_hdid, N_con * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_hnext, N_con * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_y, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_dydx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_yscal, N_con * n * sizeof(float)));

  CHECK_CUDA(hipMalloc(&d_a, N_con * n * n * sizeof(float)));  // 3D !
  CHECK_CUDA(hipMalloc(&d_dfdy, N_con * n * n * sizeof(float))); // 3D !
  CHECK_CUDA(hipMalloc(&d_indx, N_con * n * sizeof(int)));
  CHECK_CUDA(hipMalloc(&d_dfdx, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_dysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_err, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_ysav, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g1, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g2, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g3, N_con * n * sizeof(float)));
  CHECK_CUDA(hipMalloc(&d_g4, N_con * n * sizeof(float)));

  // Copy inputs to device
  CHECK_CUDA(hipMemcpy(d_x, &h_x, N_con * sizeof(float), hipMemcpyHostToDevice));
  
  CHECK_CUDA(hipMemcpy(d_y, h_y, N_con * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_dydx, h_dydx, N_con * n * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(d_yscal, h_yscal, N_con * n * sizeof(float), hipMemcpyHostToDevice));
  


}
