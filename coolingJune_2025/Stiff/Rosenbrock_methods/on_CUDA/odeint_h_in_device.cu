#include "hip/hip_runtime.h"


//----- sign_d
__device__ float sign_d(float a, float b)
{
    if (b >= 0.f)
    {
        return (a >= 0.f) ? a : -a;
    }
    else
    {
        return (a >= 0.f) ? -a : a;
    }
}


//----- odeint_d
// I can use d_y as both ystart in odeint and y in stiff. They are both the same.
__global__ void odeint_d(float *y_c, float x1, float x2, float eps, float htry, float hmin, int nvar, float *dydx_c, float *yscal_c, // y was ystart !
                              float *a_c, float *dfdy_c, int *indx_c, float *dfdx_c, float *dysav_c, float *err_c, float *ysav_c,
                              float *g1_c, float *g2_c, float *g3_c, float *g4_c, int *slot_status, int N_con, int Npart)
{

  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  
  if (idx < Npart)
  {
    
    float *y = &y_c[idx * nvar]; // Good idea, so that we do not need to do idx * nvar everywhere we have y ! Now y points to idx * nvar location !
  
    //********************************************************
    //********************************************************
    // We need to find a free slot for the particle idx !!!!!!
    //********************************************************
    //********************************************************
    //-------- find a free slot -----------
    int slot_id = -1;
    while (slot_id == -1) {
        slot_id = acquire_slot(slot_status, N_con); // N_con --> N_concurrent.
    }
    // now all of them are pointers of the start of the slot so working with them is extremely simpler now !
    float *a     = &a_c[slot_id * nvar * nvar]; // 3D
    float *dfdy  = &dfdy_c[slot_id * nvar * nvar]; // 3D
    float *yscal = &yscal_c[slot_id * nvar];
    float *dydx  = &dydx_c[slot_id * nvar];
    float *indx  = &indx_c[slot_id * nvar];
    float *dfdx  = &dfdx_c[slot_id * nvar];
    float *dysav = &dysav_c[slot_id * nvar];
    float *err   = &err_c[slot_id * nvar];
    float *ysav  = &ysav_c[slot_id * nvar];
    float *g1    = &g1_c[slot_id * nvar];
    float *g2    = &g2_c[slot_id * nvar];
    float *g3    = &g3_c[slot_id * nvar];
    float *g4    = &g4_c[slot_id * nvar];
    //-------------------------------------

    const int MAXSTP = 10000; // May need to decrease it !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
    const float TINY = 1.0e-30f;

    float x = x1;
    float h = sign_d(htry, x2 - x1);
    float hnext;

    // in y each row represents one SPH particle and each column represent an ionization fraction or ionic abundance !
    for (int nstp = 0; nstp < MAXSTP; nstp++) 
    {
      derivs(x, y, dydx, nvar, idx); // x--> scalar,  y--> N_part * nvar,   dydx--> N_part * nvar. Note: dydx is the starting pointer of the free slot!
      for (int i = 0; i < nvar; i++)
        yscal[i] = fabsf(y[i]) + fabsf(dydx[i] * h) + TINY; // Only y contains the whole 1,000,000 particles !

      if ((x + h - x2) * (x + h - x1) > 0.0f)
        h = x2 - x;

      stiff_d(y, dydx, x, h, eps, yscal, hnext, nvar,
              a, dfdy, indx, dfdx, dysav, err, ysav, g1, g2, g3, g4, idx); // n_s is n_free_slot. It will be used by 'a' and 'dfdy' in stiff !

      if ((x - x2) * (x2 - x1) >= 0.0f)
      {
        return;
      }

      if (fabsf(hnext) <= hmin) 
      {
        for (int i = 0; i < nvar; i++)
          y[i] = nanf("small_step"); // it used to be y[idx * nvar + i] but since we defined *y = &y_c[idx * nvar] we can only use y[i] instead. Great !!!
          // We should free the slot here.
          release_slot(slot_status, slot_id);
        return;
      }

      h = hnext;
    }

    for (int i = 0; i < nvar; i++)
      y[i] = nanf("too_many_steps"); // it used to be y[idx * nvar + i] but since we defined *y = &y_c[idx * nvar] we can only use y[i] instead. Great !!!
  }
}


